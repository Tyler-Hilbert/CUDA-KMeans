#include "hip/hip_runtime.h"
// KMeans in CUDA

#ifndef __KMEANS_CU__
#define __KMEANS_CU__

#include "KMeans_CUDA.h"

#include <stdio.h>
#include <random>
#include <ctime>
#include <stdexcept>
#include <string>

#include <hip/hip_runtime.h>
#include <float.h>

using namespace std;



#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}



// Computes the sum (d_sum) and count (d_count) for each of the k clusters labeled in d_centroids.
// n: number of data points
// d: number of dimensions
// k: number of clusters
// Uses shared memory of 3*k*d
__global__ void sum_and_count(
    const float *d_data,
    const float *d_centroids,
    float *d_sum,
    int *d_count,
    int n,
    int d,
    int k
) {
    // Shared memory: 
    //   0 to k*d: centroids,
    //   k*d to 2*k*d: sum,
    //   2*k*d to 3*k*d: count
    extern __shared__ float s_shared[];
    float *s_centroids = s_shared;       // Shared memory for centroids
    float *s_sum = &s_centroids[k*d];    // Shared memory for sum
    float *s_count = &s_sum[k*d];       // Shared memory for count

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Initialize shared memory
    if (tid < k * d) {
        s_centroids[tid] = d_centroids[tid];
    }
    if (tid < k) {
        s_count[tid] = 0;
    }
    if (tid < k * d) {
        s_sum[tid] = 0.0f;
    }
    __syncthreads(); // Ensure all shared memory is initialized

    if (idx < n) {
        const int idxd = idx * d;

        // Find closest centroid
        int min_class = -1;
        float dist;
        float min_dist = FLT_MAX;
        for (int c = 0; c < k; c++) {
            dist = 0;
            for (int i = 0; i < d; i++) {
                dist += pow(d_data[i+idxd] - s_centroids[i+c*d], 2);
            }
            if (dist < min_dist) {
                min_dist = dist;
                min_class = c;
            }
        }

        // Update sum and count
        atomicAdd(&s_count[min_class], 1);
        for (int i = 0; i < d; i++) {
            atomicAdd(&s_sum[i+min_class*d], d_data[i+idxd]);
        }
    }
    __syncthreads(); // Ensure all threads have finished updating sum and count

    // Write shared memory results to global memory (only one thread per centroid)
    if (tid < k) {
        atomicAdd(&d_count[tid], (int)s_count[tid]);
    }
    if (tid < k * d) {
        atomicAdd(&d_sum[tid], s_sum[tid]);
    }
}



// Updates each centroid using d_sum and d_count where the index is d * centroid number (out of K).
// d: number of dimensions
// k: number of clusters
__global__ void update_centroids(
    float *d_centroids,
    const float *d_sum,
    const int *d_count,
    int d,
    int k
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Update centroids
    if (idx < k) {
        const int idxd = idx * d;
        for (int i = 0; i < d; i++) {
          if (d_count[idx] != 0) {
            d_centroids[i+idxd] = d_sum[i+idxd] / d_count[idx];
          }
        }
    }
}



// Computes error and updates d_error
// Note this is currently only for debugging purposes as it recomputes work done in sum_and_count
// n: number of data points
// d: number of dimensions
// k: number of clusters
__global__ void calculate_error(
    const float *d_data,
    const float *d_centroids,
    float *d_error,
    int n,
    int d,
    int k
) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    if (idx < n) {
        const int idxd = idx * d;

        // Find closest centroid
        float dist;
        float min_dist = FLT_MAX;
        for (int c = 0; c < k; c++) {
            dist = 0;
            for (int i = 0; i < d; i++) {
                dist += pow(d_data[i+idxd] - d_centroids[i+c*d], 2);
            }
            if (dist < min_dist) {
                min_dist = dist;
            }
        }

        // Add error
        atomicAdd(d_error, min_dist);
    }
}



KMeans_CUDA::KMeans_CUDA(
    float *data,
    int n,
    int d,
    int k
) {
    // Seed the random number generator
    srand(static_cast<unsigned int>(time(0)));

    // CPU stack memory
    this->n = n;
    this->d = d;
    this->k = k;

    // CPU heap memory
    // Normalize data
    float *mins = new float[d];
    float *maxs = new float[d];
    for (int i = 0; i < d; i++) {
        mins[i] = data[i];
        maxs[i] = data[i];
    }
    for (int i = 0; i < n*d; i++) {
        mins[i%d] = min(mins[i%d], data[i]);
        maxs[i%d] = max(maxs[i%d], data[i]);
    }
    /*
    // Print min and max for debug
    for (int i = 0; i < d; i++) {
        printf ("min %f, max %f\n", mins[i], maxs[i]);
    }
    */
    h_data = new float[n*d];
    for (int i = 0; i < n*d; i++) {
        h_data[i] = (data[i] - mins[i%d]) / maxs[i%d];
    }
    delete[] mins;
    delete[] maxs;
    /*
    // Print data for debug
    for (int i = 0; i < n*d; i++) {
        printf ("%f ", h_data[i]);
    }
    printf ("\n");
    */

    // Centroids
    h_centroids = new float[d*k];
    for (int i = 0; i < k; i++) { // Select a datapoint for each centroid initalization
        int data_index = rand() % n;
        for (int j = 0; j < d; j++) { // Select a dimension
            h_centroids[i*d+j] = h_data[data_index*d+j];
        }
    }

    // GPU memory
    CUDA_CHECK( hipMalloc(&d_data,         n*d*sizeof(float)) );
    CUDA_CHECK( hipMalloc(&d_centroids,    k*d*sizeof(float)) );
    CUDA_CHECK( hipMalloc(&d_count,        k*sizeof(int)) );
    CUDA_CHECK( hipMalloc(&d_sum,          k*d*sizeof(float)) );

    CUDA_CHECK( hipMemcpy(d_data,      h_data,       n*d*sizeof(float), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(d_centroids, h_centroids,  k*d*sizeof(float), hipMemcpyHostToDevice) );
}



KMeans_CUDA::~KMeans_CUDA() {

    delete[] h_data;
    delete[] h_centroids;

    CUDA_CHECK( hipFree(d_data) );
    CUDA_CHECK( hipFree(d_centroids) );
    CUDA_CHECK( hipFree(d_count) );
    CUDA_CHECK( hipFree(d_sum) );
}



// Prints out the centroids
void KMeans_CUDA::print_centroids() {
    for (int i = 0; i < k; i++) {
        string s = "";
        for (int j = 0; j < d; j++) {
            s += to_string(h_centroids[i*d + j]);
            s += "  ";
        }
        s += "\n";
        printf (s.c_str());
    }
}



// Prints out predictions
void KMeans_CUDA::print_predictions() {
    for (int p = 0; p < n; p++) {
        // Find closest centroid
        int min_class = 0;
        float dist = 0;
        for (int i = 0; i < d; i++) {
            dist += pow(h_data[i+p*d] - h_centroids[i], 2);
        }

        float min_dist = dist;
        for (int c = 1; c < k; c++) {
            dist = 0;
            for (int i = 0; i < d; i++) {
                dist += pow(h_data[i+p*d] - h_centroids[i+c*d], 2);
            }
            if (dist < min_dist) {
                min_dist = dist;
                min_class = c;
            }
        }

        printf ("%i ", min_class);
    }
    printf ("\n");
}



// Runs one epoch of KMeans
void KMeans_CUDA::one_epoch() {
    // GPU setup
    CUDA_CHECK( hipMemset(d_count, 0, k*sizeof(int)) );
    CUDA_CHECK( hipMemset(d_sum,   0, k*d*sizeof(float)) );
    int threads_per_block = 32;
    int blocks1 = (n + threads_per_block - 1) / threads_per_block;
    size_t shared_mem_size = 3*k*d*sizeof(float);

    // Run kernel to get sums and counts
    sum_and_count<<<blocks1, threads_per_block, shared_mem_size>>>(d_data, d_centroids, d_sum, d_count, n, d, k);
    CUDA_CHECK( hipPeekAtLastError() );
    CUDA_CHECK( hipDeviceSynchronize() );

    // Run kernel to update centroids (calculate average)
    int blocks2 = (k + threads_per_block - 1) / threads_per_block;
    update_centroids<<<blocks2, threads_per_block>>>(d_centroids, d_sum, d_count, d, k);
    CUDA_CHECK( hipPeekAtLastError() );
    CUDA_CHECK( hipDeviceSynchronize() );

    // Copy data back to host
    CUDA_CHECK( hipMemcpy(h_centroids, d_centroids, k*d*sizeof(float), hipMemcpyDeviceToHost) );
}



float KMeans_CUDA::compute_error() {
    printf ("Note: The compute error function isn't optimized currently and is only used for debugging\n");

    // GPU setup
    float *d_error;
    float h_error = 0;
    CUDA_CHECK( hipMalloc(&d_error, sizeof(float)) );
    CUDA_CHECK( hipMemset(d_error, 0, sizeof(float)) );

    int threads_per_block = 32;
    int blocks1 = (n + threads_per_block - 1) / threads_per_block;

    // Run kernel to get sums and counts
    calculate_error<<<blocks1, threads_per_block>>>(d_data, d_centroids, d_error, n, d, k);
    CUDA_CHECK( hipPeekAtLastError() );
    CUDA_CHECK( hipDeviceSynchronize() );

    // Copy data back to host
    CUDA_CHECK( hipMemcpy(&h_error, d_error, sizeof(float), hipMemcpyDeviceToHost) );

    // Free memory
    CUDA_CHECK( hipFree(d_error) );

    return h_error;
}



#endif // __KMEANS_CU__