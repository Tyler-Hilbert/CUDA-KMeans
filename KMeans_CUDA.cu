// KMeans in CUDA

#ifndef __KMEANS_CU__
#define __KMEANS_CU__

#include <stdio.h>
#include <random>
#include <stdexcept>

#include <hip/hip_runtime.h>


#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


using namespace std;


// Computes the sum (d_sum) and count (d_count) for each of the k clusters labeled in d_centroids.
// n: number of data points
// d: number of dimensions (should be 2)
// k: number of clusters
// Uses shared memory of 3*k*d
static __global__ void sum_and_count(
    const float *d_data,
    const float *d_centroids,
    float *d_sum,
    int *d_count,
    int n,
    int d,
    int k
) {
    // Shared memory: 0 to k*d: centroids, k*d to 2*k*d: sum, 2*k*d to 3*k*d: count
    extern __shared__ float s_shared[];
    float *s_centroids = s_shared;      // Shared memory for centroids
    float *s_sum = &s_shared[k*d];    // Shared memory for sum
    int *s_count = (int*)&s_sum[k*d]; // Shared memory for count

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Initialize shared memory
    if (tid < k * d) {
        s_centroids[tid] = d_centroids[tid];
    }
    if (tid < k) {
        s_count[tid] = 0;
    }
    if (tid < k * d) {
        s_sum[tid] = 0.0f;
    }
    __syncthreads(); // Ensure all shared memory is initialized

    if (idx < n) {
        // x and y
        const int idxd = idx * d;
        float x = d_data[idxd];
        float y = d_data[idxd + 1];

        // Find closest centroid
        int min_class = 0;
        float min_dist = abs(x - s_centroids[0]) + abs(y - s_centroids[1]);
        for (int c = 1; c < k; c++) {
            const int cd = c * d;
            float dist = abs(x - s_centroids[cd]) + abs(y - s_centroids[cd + 1]);
            if (dist < min_dist) {
                min_dist = dist;
                min_class = c;
            }
        }

        // Update sum and count
        int min_class_d = min_class * d;
        atomicAdd(&s_count[min_class], 1);
        atomicAdd(&s_sum[min_class_d], x);
        atomicAdd(&s_sum[min_class_d + 1], y);
    }
    __syncthreads(); // Ensure all threads have finished updating sum and count

    // Write shared memory results to global memory (only one thread per centroid)
    if (tid < k) {
        atomicAdd(&d_count[tid], s_count[tid]);
    }
    if (tid < k * d) {
        atomicAdd(&d_sum[tid], s_sum[tid]);
    }
}


// Updates each centroid using d_sum and d_count where the index is d * centroid number (out of K).
// d: number of dimensions (should be 2)
// k: number of clusters
static __global__ void update_centroids(
    float *d_centroids,
    const float *d_sum,
    const int *d_count,
    int d,
    int k
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Update centroids
    if (idx < k) {
        const int idxd = idx * d;
        d_centroids[idxd] = d_sum[idxd] / d_count[idx];
        d_centroids[idxd+1] = d_sum[idxd+1] / d_count[idx];
    }
}

class KMeans_CUDA {
    public:
        
        KMeans_CUDA(
            float *data,
            int n, 
            int d,
            int k
        ) {

            if (d != 2) {
                throw invalid_argument("Invalid d");
            }
            if (k != 3) {
                throw invalid_argument("Invalid k");
            }

            // CPU stack memory
            this->n = n;
            this->d = d;
            this->k = k;
            this->h_data = data;

            // CPU heap memory
            h_centroids = new float[d*k];
            for (int i = 0; i < d*k; i++) {
                h_centroids[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
            }

            // GPU memory
            CUDA_CHECK( hipMalloc(&d_data,         n*d*sizeof(float)) );
            CUDA_CHECK( hipMalloc(&d_centroids,    k*d*sizeof(float)) );
            CUDA_CHECK( hipMalloc(&d_count,        k*sizeof(int)) );
            CUDA_CHECK( hipMalloc(&d_sum,          k*d*sizeof(float)) );

            CUDA_CHECK( hipMemcpy(d_data, h_data, n*d*sizeof(float), hipMemcpyHostToDevice) );
            CUDA_CHECK( hipMemcpy(d_centroids, h_centroids, k*d*sizeof(float), hipMemcpyHostToDevice) );


        }

        ~KMeans_CUDA() {
            // Note don't delete h_data since it lives in Main.cpp
            delete[] h_centroids;

            CUDA_CHECK( hipFree(d_data) );
            CUDA_CHECK( hipFree(d_centroids) );
            CUDA_CHECK( hipFree(d_count) );
            CUDA_CHECK( hipFree(d_sum) );
        }

        void printCentroids() {
            for (int i = 0; i < k; i++) {
                printf ("x %f  y %f\n", h_centroids[i*d], h_centroids[i*d+1]);
            }
        }

        // Runs one epoch of KMeans
        void one_epoch() {
            // GPU setup
            CUDA_CHECK( hipMemset(d_count, 0, k*sizeof(int)) );
            CUDA_CHECK( hipMemset(d_sum, 0, k*d*sizeof(int)) );
            int threads_per_block = 256;
            int blocks1 = (n + threads_per_block - 1) / threads_per_block;
            size_t shared_mem_size = (2*k*d*sizeof(float)) + (k*sizeof(int));

            // Run kernel to get sums and counts
            sum_and_count<<<blocks1, threads_per_block, shared_mem_size>>>(d_data, d_centroids, d_sum, d_count, n, d, k);
            CUDA_CHECK( hipPeekAtLastError() );
            CUDA_CHECK( hipDeviceSynchronize() );

            // Run kernel to update centroids (calculate average)
            int blocks2 = (k + threads_per_block - 1) / threads_per_block;
            update_centroids<<<blocks2, threads_per_block>>>(d_centroids, d_sum, d_count, d, k);
            CUDA_CHECK( hipPeekAtLastError() );
            CUDA_CHECK( hipDeviceSynchronize() );

            // Copy data back to host
            CUDA_CHECK( hipMemcpy(h_centroids, d_centroids, k*d*sizeof(int), hipMemcpyDeviceToHost) );
        }

    private:
        // Data
        float *h_data; // Data is in format { x0, y0, x1, y1, x2, y2... }. Memory stored in .cpp not class. Size n*d
        float *d_data; // Pointer to data on GPU. Size n*d

        // Learned centroids
        float *h_centroids; // Pointer to centroids in format { c0x, c0y, c1x, c1y, c2x, c2y } on heap. Size k*d
        float *d_centroids; // Pointer to centroids on GPU. Size k*d

        // Count and Sum
        float *d_sum; // Size k*d
        int *d_count; // Size k

        // Dataset 
        int n; // Number of data elements (i. e. { x0, y0, x1, y1, x2, y2} would be 3)
        int d; // Number of dimensions
        int k; // Number of clusters
};

#endif // __KMEANS_CU__